#include "hip/hip_runtime.h"
#include <thrust/sequence.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <vector>


__device__ 
bool is_case(int a) {
    bool res;
    if (a % 3 == 0) {
        res = true;
    }
    else if (a % 5 == 0) {
        res = true;
    }
    return res;
}


__global__ 
void find_cases(thrust::device_vector<int> v, thrust::device_vector<bool>& flags) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;

    if (is_case(v[i])) {
        flags[i] = true;
    }
}


int sum_multiples(thrust::host_vector<int> x, thrust::host_vector<bool> flags) {
    int n = x.size();
    int res;

    for (size_t i = 0; i < n; i++) {
        if (flags[i]) {
            res += x[i];
        }
    }
    return res;
}


int main(int argc, char* argv[]) {

    n = std::atoi(argv[1])

    // initialize host vectors
    thrust::host_vector<int> h_numbers;
    thrust::host_vector<bool> h_flags;
    thrust::sequence(h_numbers.begin(), h_numbers.end());

    // device vectors
    thrust::device_vector<int> d_numbers = h_numbers;
    thrust::device_vector<bool> d_flags;

    int grid_size = n/256;
    int block_size = 256;

    find_cases<<<grid_size, block_size>>>(d_numbers, d_flags);
    h_flags = d_flags;

    res = sum_multiples(h_numbers, h_flags);

    std::cout << res << std::endl;
}
